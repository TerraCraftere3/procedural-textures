#include "hip/hip_runtime.h"
#include "ptex/Texture.cuh"
#include "ptex/Kernel.cuh"

#include <iostream>
#include <stdlib.h>
#include <stdexcept>
#include <vector>
#include <functional>
#include <cstdint>
#include <random>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

namespace PTex
{

    // ===========================================
    // TEXTURE CLASS
    // ===========================================

    Texture::Texture(int width, int height) : m_Width(width), m_Height(height)
    {
        if (width <= 0 || height <= 0)
        {
            throw std::runtime_error("Invalid texture dimensions");
        }

        size_t size = width * height * PTEX_TEXTURE_CHANNELS;
        size_t bytes = size * sizeof(float);

        CUDA_CHECK(hipHostMalloc(&m_Data, bytes));
        memset(m_Data, 0, bytes);

        CUDA_CHECK(hipMalloc(&d_data, bytes));
        CUDA_CHECK(hipMemset(d_data, 0, bytes));

        // Verify the allocation worked
        if (!d_data)
        {
            throw std::runtime_error("Failed to allocate device memory");
        }
    }

    Texture::~Texture()
    {
        if (d_data)
            hipFree(d_data);
        if (m_Data)
            hipHostFree(m_Data);
#ifdef PTEX_USE_OPENGL
        if (m_CUDAResource)
            hipGraphicsUnregisterResource(m_CUDAResource);
        if (m_GLTexture)
            glDeleteTextures(1, &m_GLTexture);
#endif
    }

    Texture &Texture::setData(const float *data, int size)
    {
        int expectedSize = m_Width * m_Height * PTEX_TEXTURE_CHANNELS;
        if (size != expectedSize)
            throw std::runtime_error("Data size does not match texture size");

        size_t bytes = size * sizeof(float);
        CUDA_CHECK(hipMemcpy(d_data, data, bytes, hipMemcpyHostToDevice));
        return *this;
    }

    Texture &Texture::gradient(vec4 colA, vec4 colB, float angle)
    {
        dim3 blockSize(16, 16);
        dim3 gridSize((m_Width + blockSize.x - 1) / blockSize.x,
                      (m_Height + blockSize.y - 1) / blockSize.y);

        gradientKernel<<<gridSize, blockSize>>>(d_data, m_Width, m_Height, colA, colB, angle);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        return *this;
    }

    Texture &Texture::noise(float scale, float detail, float roughness, float lacunarity, float distortion)
    {
        dim3 blockSize(16, 16);
        dim3 gridSize((m_Width + blockSize.x - 1) / blockSize.x,
                      (m_Height + blockSize.y - 1) / blockSize.y);

        noiseKernel<<<gridSize, blockSize>>>(d_data, m_Width, m_Height, scale, detail, roughness, lacunarity, distortion);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        return *this;
    }

    Texture &Texture::voronoi(float scale, float detail, float roughness, float lacunarity, float smoothness)
    {
        if (scale <= 0.0f)
            scale = 1.0f;
        if (detail <= 0.0f)
            detail = 1.0f;
        if (smoothness <= 0.0f)
            smoothness = 1.0f;

        dim3 blockSize(16, 16);
        dim3 gridSize((m_Width + blockSize.x - 1) / blockSize.x,
                      (m_Height + blockSize.y - 1) / blockSize.y);

        voronoiKernel<<<gridSize, blockSize>>>(d_data, m_Width, m_Height, scale, detail, roughness, lacunarity, smoothness);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        return *this;
    }

    Texture &Texture::mix(const Texture &value, const Texture &source)
    {
        dim3 blockSize(16, 16);
        dim3 gridSize((m_Width + blockSize.x - 1) / blockSize.x,
                      (m_Height + blockSize.y - 1) / blockSize.y);

        mixKernel<<<gridSize, blockSize>>>(d_data, value.d_data, source.d_data,
                                           m_Width, m_Height, value.m_Width, value.m_Height,
                                           source.m_Width, source.m_Height);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        return *this;
    }

    Texture &Texture::add(const Texture &other)
    {
        dim3 blockSize(16, 16);
        dim3 gridSize((m_Width + blockSize.x - 1) / blockSize.x,
                      (m_Height + blockSize.y - 1) / blockSize.y);

        mathFunctionKernel<<<gridSize, blockSize>>>(d_data, other.d_data, m_Width, m_Height, other.m_Width, other.m_Height, Math::Addition{});
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        return *this;
    }

    Texture &Texture::sub(const Texture &other)
    {
        dim3 blockSize(16, 16);
        dim3 gridSize((m_Width + blockSize.x - 1) / blockSize.x,
                      (m_Height + blockSize.y - 1) / blockSize.y);

        mathFunctionKernel<<<gridSize, blockSize>>>(d_data, other.d_data, m_Width, m_Height, other.m_Width, other.m_Height, Math::Subtract{});
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        return *this;
    }

    Texture &Texture::multi(const Texture &other)
    {
        dim3 blockSize(16, 16);
        dim3 gridSize((m_Width + blockSize.x - 1) / blockSize.x,
                      (m_Height + blockSize.y - 1) / blockSize.y);

        mathFunctionKernel<<<gridSize, blockSize>>>(d_data, other.d_data, m_Width, m_Height, other.m_Width, other.m_Height, Math::Multiply{});
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        return *this;
    }

    Texture &Texture::divide(const Texture &other)
    {
        dim3 blockSize(16, 16);
        dim3 gridSize((m_Width + blockSize.x - 1) / blockSize.x,
                      (m_Height + blockSize.y - 1) / blockSize.y);

        mathFunctionKernel<<<gridSize, blockSize>>>(d_data, other.d_data, m_Width, m_Height, other.m_Width, other.m_Height, Math::Divide{});
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        return *this;
    }

    Texture &Texture::grayscale()
    {
        dim3 blockSize(16, 16);
        dim3 gridSize((m_Width + blockSize.x - 1) / blockSize.x,
                      (m_Height + blockSize.y - 1) / blockSize.y);

        grayscaleKernel<<<gridSize, blockSize>>>(d_data, m_Width, m_Height);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        return *this;
    }

    Texture &Texture::blur(float radius)
    {
        if (radius <= 0.0f)
            return *this;

        int kernelSize = int(std::ceil(radius) * 2 + 1);
        std::vector<float> kernel(kernelSize);
        float sigma = radius / 2.0f;
        float sum = 0.0f;

        int half = kernelSize / 2;
        for (int i = 0; i < kernelSize; ++i)
        {
            float x = float(i - half);
            kernel[i] = std::exp(-0.5f * (x * x) / (sigma * sigma));
            sum += kernel[i];
        }
        for (float &k : kernel)
            k /= sum;

        float *d_kernel;
        float *d_temp;
        size_t kernelBytes = kernelSize * sizeof(float);
        size_t dataBytes = m_Width * m_Height * PTEX_TEXTURE_CHANNELS * sizeof(float);

        CUDA_CHECK(hipMalloc(&d_kernel, kernelBytes));
        CUDA_CHECK(hipMalloc(&d_temp, dataBytes));
        CUDA_CHECK(hipMemcpy(d_kernel, kernel.data(), kernelBytes, hipMemcpyHostToDevice));

        dim3 blockSize(16, 16);
        dim3 gridSize((m_Width + blockSize.x - 1) / blockSize.x,
                      (m_Height + blockSize.y - 1) / blockSize.y);

        // Horizontal pass
        blurHorizontalKernel<<<gridSize, blockSize>>>(d_temp, d_data, d_kernel, m_Width, m_Height, kernelSize);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Vertical pass
        blurVerticalKernel<<<gridSize, blockSize>>>(d_data, d_temp, d_kernel, m_Width, m_Height, kernelSize);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        hipFree(d_kernel);
        hipFree(d_temp);
        return *this;
    }

    float *Texture::copy()
    {
        if (!d_data)
        {
            throw std::runtime_error("Device data is null");
        }

        size_t size = m_Width * m_Height * PTEX_TEXTURE_CHANNELS;
        size_t bytes = size * sizeof(float);

        CUDA_CHECK(hipMemcpy(m_Data, d_data, bytes, hipMemcpyDeviceToHost));
        CUDA_CHECK(hipDeviceSynchronize());

        return m_Data;
    }

    int Texture::end()
    {
#ifdef PTEX_USE_OPENGL
        if (m_GLTexture == 0)
        {
            glGenTextures(1, &m_GLTexture);
            glBindTexture(GL_TEXTURE_2D, m_GLTexture);
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
            // Allocate GPU storage
            glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, m_Width, m_Height, 0, GL_RGBA, GL_FLOAT, nullptr);

            // Register with CUDA
            CUDA_CHECK(hipGraphicsGLRegisterImage(&m_CUDAResource, m_GLTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
        }

        // Map CUDA resource
        CUDA_CHECK(hipGraphicsMapResources(1, &m_CUDAResource, 0));
        hipArray_t array;
        CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(&array, m_CUDAResource, 0, 0));

        // Copy your device data into the GL texture
        CUDA_CHECK(hipMemcpy2DToArray(
            array,
            0, 0,
            d_data,
            m_Width * PTEX_TEXTURE_CHANNELS * sizeof(float),
            m_Width * PTEX_TEXTURE_CHANNELS * sizeof(float),
            m_Height,
            hipMemcpyDeviceToDevice));

        CUDA_CHECK(hipGraphicsUnmapResources(1, &m_CUDAResource, 0));

        return m_GLTexture;
#else
#pragma message("No OpenGL Support for Texture.cu!!!");
        return 0;
#endif
    }

    const float *Texture::getData() const
    {
        return m_Data;
    }

    int Texture::getTextureID() const
    {
        return m_GLTexture;
    }

}